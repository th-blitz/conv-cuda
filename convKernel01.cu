#include "hip/hip_runtime.h"

// copyright (c) th-blitz (https://github.com/th-blitz) 2024

#include <stdint.h>
#include <stdio.h>
#include "convKernel.h"

#define FILTER_HEIGHT 3
#define FILTER_WIDTH 3
#define INPUT_CHANNELS 3 
#define OUTPUT_CHANNELS 64
#define BLOCK_SIZE 16

__global__ void ConvForward(const struct Filter filter, const struct Data_Tensor input_tensor, struct Data_Tensor output_tensor) {
  
    int32_t thread_x = threadIdx.x;
    int32_t thread_y = threadIdx.y;
    int32_t thread_z = threadIdx.z;

    int32_t x_stride = blockDim.x;
    int32_t y_stride = blockDim.y;
    int32_t z_stride = blockDim.z;

    __shared__ double FilterTile[OUTPUT_CHANNELS][INPUT_CHANNELS][FILTER_HEIGHT][FILTER_WIDTH];
    __shared__ double InputTile[INPUT_CHANNELS][BLOCK_SIZE + FILTER_HEIGHT - 1][BLOCK_SIZE + FILTER_WIDTH - 1];

    int32_t grid_size_x = output_tensor.channels / gridDim.x;
    int32_t grid_offset_x = blockIdx.x * grid_size_x;

    int32_t grid_size_y = output_tensor.height / gridDim.y;
    int32_t grid_offset_y = blockIdx.y * grid_size_y;
    
    int32_t grid_size_z = output_tensor.width / gridDim.z;
    int32_t grid_offset_z = blockIdx.z * grid_size_z;



    for (int32_t o_channel = thread_x + grid_offset_x; o_channel < grid_offset_x + grid_size_x; o_channel += x_stride) {
        for (int32_t o_row = thread_y + grid_offset_y; o_row < grid_offset_y + grid_size_y; o_row += y_stride) {
            for (int32_t o_col = thread_z + grid_offset_z; o_col < grid_offset_z + grid_size_z; o_col += z_stride) {

                if (thread_y < 3 && thread_z < 3) {
                    #pragma unroll
                    for (int32_t i_channel = 0; i_channel < INPUT_CHANNELS; i_channel += 1) {
                        FilterTile[o_channel][i_channel][thread_y][thread_z] = filter.elements[
                            o_channel * (filter.input_channels * filter.height * filter.width) +
                            i_channel * (filter.height * filter.width) +
                            (filter.height - 1 - thread_y) * filter.width +
                            (filter.width - 1 - thread_z)
                        ];
                    }
                }

                double* input_val;
                if (thread_x < INPUT_CHANNELS) {

                    input_val = &input_tensor.elements[
                        thread_x * (input_tensor.true_height * input_tensor.true_width) + 
                        (o_row) * (input_tensor.true_width) +
                        (o_col) 
                    ]; 

                    InputTile[thread_x][thread_y][thread_z] = input_val[0]; 

                    if (thread_y == BLOCK_SIZE - 1) {
                        InputTile[thread_x][thread_y + 1][thread_z] = input_val[input_tensor.true_width];                       
                        InputTile[thread_x][thread_y + 2][thread_z] = input_val[2 * input_tensor.true_width];                       
                    }

                    if (thread_z == BLOCK_SIZE - 1) {
                        InputTile[thread_x][thread_y][thread_z + 1] = input_val[1];                       
                        InputTile[thread_x][thread_y][thread_z + 2] = input_val[2];                       
                    }

                    if (thread_y == (BLOCK_SIZE - 1) && thread_z == (BLOCK_SIZE - 1)) {
                        InputTile[thread_x][thread_y + 1][thread_z + 1] = input_val[input_tensor.true_width + 1];                       
                        InputTile[thread_x][thread_y + 1][thread_z + 2] = input_val[input_tensor.true_width + 2]; 
                        InputTile[thread_x][thread_y + 2][thread_z + 1] = input_val[2 * input_tensor.true_width + 1];                       
                        InputTile[thread_x][thread_y + 2][thread_z + 2] = input_val[2 * input_tensor.true_width + 2];                       
                    } 
                }
                                      
                __syncthreads();

                double output_element_value = 0.0;

                for (int32_t i_channel = 0; i_channel < INPUT_CHANNELS; i_channel += 1) {
                    for (int32_t row = 0; row < FILTER_HEIGHT; row += 1) {
                        for (int32_t col = 0; col < FILTER_WIDTH; col += 1) {

                            output_element_value += FilterTile[o_channel][i_channel][row][col] * 
                                InputTile[i_channel][thread_y + row][thread_z + col];

                        }
                    }
                }

                output_tensor.elements[
                        o_channel * (output_tensor.true_height * output_tensor.true_width) +
                        o_row * (output_tensor.true_width) +
                        o_col 
                ] = output_element_value;

                __syncthreads();
            }
        }
    }
 
}
