
#include <hip/hip_runtime.h>
// vecAddKernel00.cu
// For ECE-GY 9143 - High Performance Computing for Machine Learning
// Instructor: Parijat Dubey
// Based on code from the CUDA Programming Guide

// This Kernel adds two Vectors A and B in C on GPU
// without using coalesced memory access.
__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;  // Calculate global index for each thread
    int stride = blockDim.x * gridDim.x;  // Total number of threads

	// contigious memory accesses     
    for(int i = index; i < (N * stride); i += stride){
        C[i] = A[i] + B[i];
    }

    __syncthreads();
}
//
//__global__ void AddVectors(const float* A, const float* B, float* C, int N)
//{
//    int blockStartIndex  = blockIdx.x * blockDim.x * N;
//    int threadStartIndex = blockStartIndex + (threadIdx.x * N);
//    int threadEndIndex   = threadStartIndex + N;
//    int i;
//
//    for( i=threadStartIndex; i<threadEndIndex; ++i ){
//        C[i] = A[i] + B[i];
//    }
//}
