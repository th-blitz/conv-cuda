#include "hip/hip_runtime.h"
#include "matmultKernel.h"
#include <stdio.h>




void print_sub_mat(float* sub, int size) {
    for ( int x = 0; x < size; x++ ) {
        for ( int y = 0; y < size; y++) {
            printf("%f ", sub[x * size + y]);
        }
        printf("\n");
    }
}


__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){

	float *Asub, *Bsub, *Csub;
	
	int thread_row = threadIdx.y;
	int thread_col = threadIdx.x;
	int block_row = blockIdx.y;
	int block_col = blockIdx.x;
	
	Csub = &C.elements[C.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * block_col];

	// To store the 4 output values:
	float Cvalues[] = {0, 0, 0, 0, 0, 0, 0, 0};
	// Offsets arrays to offset thread indices while performing matrix multiplications over 32x32 sub matrices.
	int a_row_offset[] = {0, 0, 0, 0, 16, 16, 16, 16};
	int a_col_offset[] = {0, 16, 0, 16, 0, 16, 0, 16};
	int b_row_offset[] = {0, 16, 0, 16, 0, 16, 0, 16};
	int b_col_offset[] = {0, 0, 16, 16, 0, 0, 16, 16};
	
	for (int m = 0;  m < (A.width / FOOTPRINT_SIZE); ++m) {

		Asub = &A.elements[A.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * m];
		Bsub = &B.elements[B.stride * FOOTPRINT_SIZE * m + FOOTPRINT_SIZE * block_col];
	
		// Each thread calculates outputs for 4 values across 32x32 sub matrices.
#pragma unroll
    	for (int i = 0; i < 8; i++) {
			
			// load a 16x16 block to shared memory
    		__shared__ float shared_A[BLOCK_SIZE][BLOCK_SIZE];
			__shared__ float shared_B[BLOCK_SIZE][BLOCK_SIZE];
			
			shared_A[thread_row][thread_col] = Asub[(thread_row + a_row_offset[i]) * A.stride + (thread_col + a_col_offset[i])];
			shared_B[thread_row][thread_col] = Bsub[(thread_row + b_row_offset[i]) * B.stride + (thread_col + b_col_offset[i])];
			
			__syncthreads();
			
			// perform matrix multiplications over 16x16 blocks
#pragma unroll
			for(int e=0; e<BLOCK_SIZE; ++e) {
			   Cvalues[i] += shared_A[thread_row][e] * shared_B[e][thread_col];
			} 
			
			__syncthreads();
		
    	}
	}

	// assign all 4 output values:
	Csub[thread_row * C.stride + thread_col] = Cvalues[0] + Cvalues[1];
	Csub[thread_row * C.stride + (thread_col + 16)] = Cvalues[2] + Cvalues[3];
	Csub[(thread_row + 16) * C.stride + thread_col] = Cvalues[4] + Cvalues[5];
	Csub[(thread_row + 16) * C.stride + (thread_col + 16)] = Cvalues[6] + Cvalues[7];
}

