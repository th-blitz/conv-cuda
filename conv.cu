#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "timer.h"
#include "convKernel.h"
#include <hipDNN.h>
#include <hip/hip_runtime.h>


#define BLOCK_SIZE 16

#define OUTPUT_CHANNELS 64
#define INPUT_CHANNELS 3
#define FILTER_HEIGHT 3
#define FILTER_WIDTH 3

#define HEIGHT 1024
#define WIDTH 1024




struct Filter MakeFilters(int32_t output_channels, int32_t input_channels, int32_t filter_height, int32_t filter_width) {

    struct Filter filter;
    filter.output_channels = output_channels;
    filter.input_channels = input_channels;
    filter.height = filter_height;
    filter.width = filter_width;

    filter.elements = (double*)malloc(output_channels * input_channels * filter_height * filter_width * sizeof(double));

    for (int32_t o_channel = 0; o_channel < output_channels; o_channel++) {
        for (int32_t i_channel = 0; i_channel < input_channels; i_channel++) {
            for (int32_t row = 0; row < filter_height; row++) {
                for (int32_t col = 0; col < filter_width; col++) {
                    double element_value = (o_channel + i_channel) * (row + col);
                    filter.elements[
                        o_channel * (input_channels * filter_height * filter_width) +
                        i_channel * (filter_height * filter_width) +
                        row * (filter_width) +
                        col
                    ] = element_value;
                }
            }
        }
    }

    return filter;
}

double* CudnnDeviceFilter(struct Filter filter) {
   
    double* d_cudnn_filter;
    double cudnn_filter [64][3][3][3];

    for (int32_t o_channel = 0; o_channel < filter.output_channels; o_channel++) {
        for (int32_t i_channel = 0; i_channel < filter.input_channels; i_channel++) {
            for (int32_t row = 0; row < filter.height; row++) {
                for (int32_t col = 0; col < filter.width; col++) {
                    cudnn_filter[o_channel][i_channel][row][col] = filter.elements[
                        o_channel * (filter.input_channels * filter.height * filter.width) +
                        i_channel * (filter.height * filter.width) +
                        row * (filter.width) +
                        col
                    ];
                }
            }
        }
    }
    
    size_t size = 64 * 3 * 3 * 3 * sizeof(double); 
    hipMalloc((void**) &d_cudnn_filter, size);
    hipMemcpy(d_cudnn_filter, cudnn_filter, size, hipMemcpyHostToDevice);

    return d_cudnn_filter;
}
void PrintFilter(struct Filter filter, const char* name) {

    int32_t output_channels = filter.output_channels;
    int32_t input_channels = filter.input_channels;
    int32_t height = filter.height;
    int32_t width = filter.width;
    
    printf("\n%s \n",name);
    printf("output_channels : %d \n", output_channels);
    printf("input_channels : %d \n", input_channels);
    printf("height : %d \n", height);
    printf("width : %d \n", width);
    
    for (int32_t o_channel = 0; o_channel < output_channels; o_channel++) {
        printf("\nOutput Channel : %d\n", o_channel);
        for (int32_t i_channel = 0; i_channel < input_channels; i_channel++) {
            printf("--- Input Channel : %d\n", i_channel);
            for (int32_t row = 0; row < height; row++) {
                printf("   | ");
                for (int32_t col = 0; col < width; col++) {
                    double element_value = filter.elements[
                        o_channel * (input_channels * height * width) +
                        i_channel * (height * width) +
                        row * (width) +
                        col
                    ];
                    printf("%lf ", element_value); 
                }
                printf(";\n");
            }
        }
    }
   
}

struct Data_Tensor MakeTensors(int32_t channels, int32_t height, int32_t width, int32_t padding) {
    
    struct Data_Tensor tensor;
    
    int32_t true_height = height + (2 * padding);
    int32_t true_width = width + (2 * padding);

    tensor.channels = channels;
    tensor.height = height;
    tensor.width = width;
    tensor.padding = padding;
    tensor.true_height = true_height;
    tensor.true_width = true_width;
    tensor.elements = (double*)malloc(channels * true_height * true_width * sizeof(double));

    for (int32_t channel = 0; channel < channels; channel++) {
        for (int32_t row = -1 * padding; row < height + padding; row++) {
            for (int32_t col = -1 * padding; col < width + padding; col++) {

                double element = (channel) * (row + col);  
                if (row < 0 || col < 0 || row >= height || col >= width) {
                    element = 0.0;
                }
                tensor.elements[(channel * (true_height * true_width)) + (true_width * (row + padding)) + (col + padding)] = element;

            }
        }
    }
        
    return tensor;
}

void PrintTensor(struct Data_Tensor tensor, const char* name) {

    int32_t channels = tensor.channels;
    int32_t height = tensor.height;
    int32_t width = tensor.width;
    int32_t padding = tensor.padding;
    int32_t true_height = tensor.true_height;
    int32_t true_width = tensor.true_width;

    printf("\n%s \n",name);
    printf("channels : %d \n", channels);
    printf("height : %d \n", height);
    printf("width : %d \n", width);
    printf("padding : %d \n\n", padding);
    
    int32_t row, col;
    for (int32_t channel = 0; channel < channels; channel++) {
        printf("channel : %d\n", channel);
        for (row = -padding; row < height + padding; row++) {
            for (col = -padding; col < width + padding; col++) {

                double element = tensor.elements[(channel * (true_height * true_width)) + (true_width * (row + padding)) + (col + padding)];
                printf("%lf ", element);

            }
            printf(";\n");
        }
    }

}

struct Data_Tensor CopyTensorToDevice(struct Data_Tensor tensor, bool copy) {
    
    struct Data_Tensor device_tensor;

    device_tensor.channels = tensor.channels;
    device_tensor.height = tensor.height;
    device_tensor.width = tensor.width;
    device_tensor.padding = tensor.padding;
    device_tensor.true_height = tensor.true_height;
    device_tensor.true_width = tensor.true_width;
    size_t size = tensor.channels * tensor.true_height * tensor.true_width * sizeof(double); 
    hipMalloc((void**) &device_tensor.elements, size);
    if (copy)
        hipMemcpy(device_tensor.elements, tensor.elements, size, hipMemcpyHostToDevice);
    return device_tensor;
}

struct Filter CopyFilterToDevice(struct Filter filter, bool copy) {
    
    struct Filter device_filter;

    device_filter.input_channels = filter.input_channels;
    device_filter.output_channels = filter.output_channels;
    device_filter.height = filter.height;
    device_filter.width = filter.width;
    size_t size = filter.output_channels * filter.input_channels * filter.height * filter.width * sizeof(double); 
    hipMalloc((void**)&device_filter.elements, size);
    hipMemcpy(device_filter.elements, filter.elements, size, hipMemcpyHostToDevice);
    return device_filter;
}

double PrintSampleResult(struct Data_Tensor input_tensor, struct Data_Tensor output_tensor, struct Filter filter, bool print) {
  
    double checksum = 0.0;

    for (int32_t o_channel = 0; o_channel < output_tensor.channels; o_channel += 1) {
        for (int32_t o_row = 0; o_row < output_tensor.true_height; o_row += 1) {
            for (int32_t o_col = 0; o_col < output_tensor.true_width; o_col += 1) {
                double output_element_value = 0.0;
                for (int32_t i_channel = 0; i_channel < filter.input_channels; i_channel += 1) {
                    for (int32_t row = 0; row < filter.height; row += 1) {
                        for (int32_t col = 0; col < filter.width; col += 1) {
                            output_element_value += filter.elements[
                                o_channel * (filter.input_channels * filter.height * filter.width) +
                                i_channel * (filter.height * filter.width) +
                                (filter.height - 1 - row) * filter.width +
                                (filter.width - 1 - col)
                            ] * input_tensor.elements[
                                i_channel * (input_tensor.true_height * input_tensor.true_width) + 
                                (o_row + row) * (input_tensor.true_width) +
                                (o_col + col) 
                            ];
                        }
                    }
                }
                output_tensor.elements[
                    o_channel * (output_tensor.true_height * output_tensor.true_width) +
                    o_row * (output_tensor.true_width) +
                    o_col
                ] = output_element_value;
                checksum += output_element_value;
            }
        }
    }
    if (print)
        PrintTensor(output_tensor, "print sample output on CPU : "); 
    return checksum;
}

double GenerateCheckSum(struct Data_Tensor output_tensor) {
    double checksum = 0.0;
    for (int32_t o_channel = 0; o_channel < output_tensor.channels; o_channel += 1) {
        for (int32_t o_row = 0; o_row < output_tensor.true_height; o_row += 1) {
            for (int32_t o_col = 0; o_col < output_tensor.true_width; o_col += 1) {
                checksum += output_tensor.elements[
                    o_channel * (output_tensor.true_height * output_tensor.true_width) +
                    o_row * (output_tensor.true_width) +
                    o_col
                ];
            }
        }
    }
    return checksum;
}

void PrintTime(double checksum, double time, uint32_t output_height, uint32_t output_width) {
    double nFlops = (double)( 2 * OUTPUT_CHANNELS * output_height * output_width * INPUT_CHANNELS * FILTER_HEIGHT * FILTER_WIDTH);
    double nFlopsPerSec = nFlops / time;
    double nGFlopsPerSec = nFlopsPerSec*1e-9;
    printf( "Checksum: %lf, Time: %lf (milli sec), GFlopsS: %lf\n",
              checksum, time * 1000, nGFlopsPerSec);
 }

int main(int argc, char* argv[]) {
    
    printf("Number of arguments: %d \n", argc);
    printf("Arguments:\n");
    for (int i = 0; i < argc; i++) {
        printf("argv[%d]: %s\n", i, argv[i]);
    }

   //H = 1024, W = 1024, C = 3, FW = 3, F H = 3, K = 64 
    
//    sscanf(argv[1], "%d", &height);
//    sscanf(argv[1], "%d", &width);
//    sscanf(argv[1], "%d", &input_channels);
//    sscanf(argv[1], "%d", &filter_width);
//    sscanf(argv[1], "%d", &filter_height);
//  
    uint32_t padding = 1; 
    uint32_t input_channels = INPUT_CHANNELS;
    uint32_t output_channels = OUTPUT_CHANNELS;
    uint32_t height = HEIGHT;
    uint32_t width = WIDTH;
    uint32_t filter_height = FILTER_HEIGHT;
    uint32_t filter_width = FILTER_WIDTH;

    uint32_t output_height;
    uint32_t output_width;

    output_height = (
        (height - filter_height + (2*padding)) / 1 // stride = 1 
    ) + 1;

    output_width = (
        (width - filter_width + (2*padding)) / 1 // stride = 1 
    ) + 1;

    dim3 dimGrid(4, 16, 16);
    dim3 dimBlock(4, BLOCK_SIZE, BLOCK_SIZE);

   // printf("Input dim : (%d, %d, %d) \n", input_channels, height, width);
   // printf("Output dim ( dimGrid ) : (%d, %d, %d) \n", output_channels, output_height, output_width);
   // printf("Filter dim ( dimBlock ) : (%d, %d, %d, %d) \n", output_channels, input_channels, filter_height, filter_width);
   
    struct Data_Tensor input_tensor = MakeTensors(input_channels, height, width, padding);
    struct Data_Tensor output_tensor = MakeTensors(output_channels, output_height, output_width, 0);
    struct Filter filter = MakeFilters(output_channels, input_channels, filter_height, filter_width); 
    //PrintTensor(input_tensor, "test input tensor :");
    // PrintFilter(filter, "test filter : "); 

    size_t size = output_tensor.channels * output_tensor.true_height * output_tensor.true_width * sizeof(double); 

    struct Data_Tensor device_input_tensor = CopyTensorToDevice(input_tensor, true);
    struct Data_Tensor device_output_tensor = CopyTensorToDevice(output_tensor, false);
    struct Filter device_filter = CopyFilterToDevice(filter, true);


/////////////////////////   Conv Forward 00 Kernel ////////////////////////////////
    // Invoke kernel for warm up
    ConvForward00<<<dimGrid, dimBlock>>>(device_filter, device_input_tensor, device_output_tensor);      
    // Synchronize to make sure everyone is done in the warmup.
    hipDeviceSynchronize();
    // Set up timer
    initialize_timer();
    start_timer();
    // Invoke kernel for real
    ConvForward00<<<dimGrid, dimBlock>>>(device_filter, device_input_tensor, device_output_tensor);      
    // Synchronize to make sure everyone is done.
    hipDeviceSynchronize() ;
    // Compute and report the timing results
    stop_timer();
    double ConvForward00_time = elapsed_time();
    hipMemcpy(output_tensor.elements, device_output_tensor.elements, size, hipMemcpyDeviceToHost);
    // PrintTensor(output_tensor, "test output tensor : ");
    double ConvForward00_checksum = GenerateCheckSum(output_tensor);
    PrintTime(ConvForward00_checksum, ConvForward00_time, output_height, output_width);

/////////////////////////   Conv Forward 01 Kernel ////////////////////////////////
    // Invoke kernel for warm up
    ConvForward01<<<dimGrid, dimBlock>>>(device_filter, device_input_tensor, device_output_tensor);      
    // Synchronize to make sure everyone is done in the warmup.
    hipDeviceSynchronize();
    // Set up timer
    initialize_timer();
    start_timer();
    // Invoke kernel for real
    ConvForward01<<<dimGrid, dimBlock>>>(device_filter, device_input_tensor, device_output_tensor);      
    // Synchronize to make sure everyone is done.
    hipDeviceSynchronize() ;
    // Compute and report the timing results
    stop_timer();
    double ConvForward01_time = elapsed_time();
    hipMemcpy(output_tensor.elements, device_output_tensor.elements, size, hipMemcpyDeviceToHost);
    // PrintTensor(output_tensor, "test output tensor : ");
    double ConvForward01_checksum = GenerateCheckSum(output_tensor);
    PrintTime(ConvForward01_checksum, ConvForward01_time, output_height, output_width);


/////////////////////////   Cudnn Kernel ////////////////////////////////

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    hipdnnStatus_t status;

    hipdnnTensorDescriptor_t input_descriptor, output_descriptor;
    hipdnnFilterDescriptor_t filter_descriptor;
    int NumOfDims_Tensor = 4;
    const int InputTensorDim[] = {1, 3, 1024, 1024};
    const int InputTensorStride[] = {3 * 1024 * 1024, 1024 * 1024, 1024, 1};
    
    const int OutputTensorDim[] = {1, 64, 1024, 1024};
    const int OutputTensorStride[] = {64 * 1024 * 1024, 1024 * 1024, 1024, 1};

    const int FilterTensorDim[] = {64, 3, 3, 3};

    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnSetTensorNdDescriptor(input_descriptor, HIPDNN_DATA_DOUBLE, NumOfDims_Tensor, InputTensorDim, InputTensorStride);
    
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnSetTensorNdDescriptor(output_descriptor, HIPDNN_DATA_DOUBLE, NumOfDims_Tensor, OutputTensorDim, OutputTensorStride);
    
    
    hipdnnCreateFilterDescriptor(&filter_descriptor);
    status = hipdnnSetFilterNdDescriptor(filter_descriptor, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, 4, FilterTensorDim);  
    
    if (status != HIPDNN_STATUS_SUCCESS) { 
       printf("Error: Failed to create tensor descriptor: %s\n", hipdnnGetErrorString(status)); 
    }

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    hipdnnCreateConvolutionDescriptor(&convolution_descriptor);
    status = hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                           /*pad_height=*/1,
                                           /*pad_width=*/1,
                                           /*vertical_stride=*/1,
                                           /*horizontal_stride=*/1,
                                           /*dilation_height=*/1,
                                           /*dilation_width=*/1,
                                           /*mode=*/HIPDNN_CONVOLUTION,
                                           /*computeType=*/HIPDNN_DATA_DOUBLE);
    if (status != HIPDNN_STATUS_SUCCESS) { 
       printf("Error: Failed to create tensor descriptor: %s\n", hipdnnGetErrorString(status)); 
    }


    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    status = hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                        input_descriptor,
                                        filter_descriptor,
                                        convolution_descriptor,
                                        output_descriptor,
                                        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                        /*memoryLimitInBytes=*/0,
                                        &convolution_algorithm);
    if (status != HIPDNN_STATUS_SUCCESS) { 
       printf("Error: Failed to create tensor descriptor: %s\n", hipdnnGetErrorString(status)); 
    }


    size_t workspace_bytes = 0;
    status = hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                            input_descriptor,
                                            filter_descriptor,
                                            convolution_descriptor,
                                            output_descriptor,
                                            convolution_algorithm,
                                            &workspace_bytes);
    if (status != HIPDNN_STATUS_SUCCESS) { 
       printf("Error: Failed to create tensor descriptor: %s\n", hipdnnGetErrorString(status)); 
    }


    // printf("workspace bytes : %ld\n", workspace_bytes);
    void* device_workspace_bytes;
    hipMalloc(&device_workspace_bytes, workspace_bytes);

    initialize_timer();
    start_timer();
    
    const double alpha = 1, beta = 0;
    status = hipdnnConvolutionForward(cudnn,
                            &alpha,
                            input_descriptor,
                            device_input_tensor.elements,
                            filter_descriptor,
                            device_filter.elements, 
                            convolution_descriptor,
                            convolution_algorithm,
                            device_workspace_bytes,
                            workspace_bytes,
                            &beta,
                            output_descriptor,
                            device_output_tensor.elements);

    hipDeviceSynchronize();
    stop_timer();
    double cudnn_conv_time = elapsed_time();

    if (status != HIPDNN_STATUS_SUCCESS) { 
       printf("Error: Failed to create tensor descriptor: %s\n", hipdnnGetErrorString(status)); 
    }

    hipMemcpy(output_tensor.elements, device_output_tensor.elements, size, hipMemcpyDeviceToHost);
    double cudnn_conv_checksum = GenerateCheckSum(output_tensor);
    PrintTime(cudnn_conv_checksum, cudnn_conv_time, output_height, output_width);
    // PrintTensor(output_tensor, "cudnn output : ");


    double true_checksum = PrintSampleResult(input_tensor, output_tensor, filter, false);
    printf("true checksum : %lf ( calculated on cpu ) \n", true_checksum);

    hipFree(device_input_tensor.elements);
    hipFree(device_output_tensor.elements);
    hipFree(device_filter.elements);
    hipFree(device_workspace_bytes);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);
    
    free(input_tensor.elements);
    free(output_tensor.elements);
    free(filter.elements);


    return 0;

}

    
