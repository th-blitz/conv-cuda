#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.h"
#include "vecaddKernel.h"


float* array_a;
float* array_b;
float* array_c;


int main(int argc, char* argv[]) {

    //printf("Number of arguments: %d \n", argc);
    //printf("Arguments:\n");
    //for (int i = 0; i < argc; i++) {
    //    printf("argv[%d]: %s\n", i, argv[i]);
    //}
    
    int k;
    int grid_dim = 1;
    int block_dim = 1;
    sscanf(argv[1], "%d", &k);
    sscanf(argv[2], "%d", &grid_dim);
    sscanf(argv[3], "%d", &block_dim);

    int size = k * 1000000;
    int values_per_thread = size / (grid_dim * block_dim);

    printf("Total vector size : %d\n", size);

    dim3 dimGrid(grid_dim);
    dim3 dimBlock(block_dim);

    size_t vec_size = size * sizeof(float);

    hipMallocManaged(&array_a, vec_size);
    hipMallocManaged(&array_b, vec_size);
    hipMallocManaged(&array_c, vec_size);

    for (int i = 0; i < size; ++i) {
        array_a[i] = (float)i;
        array_b[i] = (float)(size - i);
        array_c[i] = 0.0;
    }


    AddVectors<<< dimGrid, dimBlock >>>(array_a, array_b, array_c, size);
    hipDeviceSynchronize();

    initialize_timer();
    start_timer();

    AddVectors<<< dimGrid, dimBlock >>>(array_a, array_b, array_c, size);
    hipDeviceSynchronize();

    stop_timer();
    double time = elapsed_time();

    int nFlops = size;
    double nFlopsPerSec = nFlops/time;
    double nGFlopsPerSec = nFlopsPerSec*1e-9;

	// Compute transfer rates.
    int nBytes = 3 * 4 * size; // 2N words in, 1N word out
    double nBytesPerSec = nBytes/time;
    double nGBytesPerSec = nBytesPerSec*1e-9;

	// Report timing data.
    printf( "Time: %lf (sec), GFlopsS: %lf, GBytesS: %lf\n", 
             time, nGFlopsPerSec, nGBytesPerSec);

    int i;
    for (i = 0; i < size; ++i) {
        float val = array_c[i];
        //printf("%d : %f\n", i, val);
        if (fabs(val - size) > 1e-5)
            break;
    }
    printf("Test %s \n", (i == size) ? "PASSED" : "FAILED");


    hipFree(array_a);
    hipFree(array_b);
    hipFree(array_c);

    return 0;
}

